#include "hip/hip_runtime.h"
/*
    Copyright (C) 2011  Abhinav Jauhri (abhinav.jauhri@gmail.com), Carnegie Mellon UniversithreadIdx.y - Silicon Valley 

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANthreadIdx.y; without even the implied warranthreadIdx.y of
    MERCHANTABILIthreadIdx.y or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#define TILE_WIDTH 32
#define TILE_WIDTH_SHIFT 5
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
#include "stdio.h"
#include <sys/time.h>
//#define OUTPUT_TIME 
//#define TILE_WIDTH 2

namespace cuda
{

__device__ void saxpy( float a, float *b, float *c )
{
    c[0] += a*b[0];
    c[1] += a*b[1];
    c[2] += a*b[2];
    c[3] += a*b[3];
    c[4] += a*b[4];
    c[5] += a*b[5];
    c[6] += a*b[6];
    c[7] += a*b[7];
    c[8] += a*b[8];
    c[9] += a*b[9];
    c[10] += a*b[10];
    c[11] += a*b[11];
    c[12] += a*b[12];
    c[13] += a*b[13];
    c[14] += a*b[14];
    c[15] += a*b[15];
    c[16] += a*b[16];
    c[17] += a*b[17];
    c[18] += a*b[18];
    c[19] += a*b[19];
    c[20] += a*b[20];
    c[21] += a*b[21];
    c[22] += a*b[22];
    c[23] += a*b[23];
    c[24] += a*b[24];
    c[25] += a*b[25];
    c[26] += a*b[26];
    c[27] += a*b[27];
    c[28] += a*b[28];
    c[29] += a*b[29];
    c[30] += a*b[30];
    c[31] += a*b[31];
}

#define STEP 32
#define GROUP 4
//thread : (STEP,GROUP)

 __global__ void matrixMultiply_1024( const float *A, const float *B, float* C)
{
    const int inx = threadIdx.x;
    const int iny = threadIdx.y;
    const int ibx = blockIdx.x * STEP * GROUP;
    const int iby = blockIdx.y * STEP;
    const int id = inx + iny * STEP;

    A += ibx + id;
    B += inx + ( iby + iny) * 1024 ;
    C += ibx + id  + ( iby * 1024 );
    
//    if (blockIdx.x!=2||blockIdx.y!=3)
//        return;
//    printf("(%d,%d:%d,%d):A+=%d B+=%d C+=%d \n", blockIdx.x , blockIdx.y, inx,iny,
//             ibx + id, inx + ( iby + iny) * ldb ,ibx + id  + ( iby * ldc ));
    
    //const float *Blast = B + dim;

    float c[STEP] = {0};

    __shared__ float bs[STEP][STEP + 1];
    //do
#pragma unroll
    for (int t=0;t<1024/STEP;t++)
    {
//#pragma unroll
        //for( int i = 0; i < STEP; i += GROUP*2 ){
        //    bs[inx][iny+i]  = B[i*dim];
        //}
            bs[inx][iny]  = B[0];
            bs[inx][iny+GROUP]  = B[(GROUP)*1024];
            bs[inx][iny+2*GROUP]  = B[(2*GROUP)*1024];
            bs[inx][iny+3*GROUP]  = B[(3*GROUP)*1024];
            bs[inx][iny+4*GROUP]  = B[(4*GROUP)*1024];
            bs[inx][iny+5*GROUP]  = B[(5*GROUP)*1024];
            bs[inx][iny+6*GROUP]  = B[(6*GROUP)*1024];
            bs[inx][iny+7*GROUP]  = B[(7*GROUP)*1024];


        __syncthreads();

#pragma unroll
        for( int i = 0; i < STEP; i++, A += 1024 ){
            saxpy( A[0], &bs[i][0], c ); 
        }
        
        B += STEP;

        
        __syncthreads();
    } //while( B < Blast );

    for( int i = 0; i < STEP; i++, C += 1024 )
        C[0] = c[i]; 
}	



 __global__ void matrixMultiply_1000( const float *A, const float *B, float* C)
{
    const int inx = threadIdx.x;
    const int iny = threadIdx.y;
    const int ibx = blockIdx.x * STEP * GROUP;
    const int iby = blockIdx.y * STEP;
    const int id = inx + iny * STEP;

    A += ibx + id;
    B += inx + ( iby + iny) * 1000 ;
    C += ibx + id  + ( iby * 1000 );
    
    //printf("[%d,%d] ",blockIdx.x,blockIdx.y);

    float c[STEP] = {0};

    __shared__ float bs[STEP][STEP + 1];
    
//    if (threadIdx.x==0 && threadIdx.y==0)
//                printf("ix:%d iy:%d \n",inx,iny);
    //do
#pragma unroll
    for (int t=0;t<(1000-1)/STEP+1;t++)
    {
#pragma unroll
        for( int i = 0; i < STEP; i += GROUP ){
            if ((t*STEP+inx>=1000) || (iby+iny+i>=1000))
                bs[inx][iny+i] = 0;
            else {
                bs[inx][iny+i] = B[i*1000];
  //              printf("i:%d ix:%d iy:%d \n",i,inx,iny);
            }
        }


        __syncthreads();

  if (ibx+id<1000){
#pragma unroll
        for( int i = 0; i < STEP; i++, A += 1000 ){
            if (t*STEP+i<1000) {
                saxpy( A[0], &bs[i][0], c ); 
                //printf("i:%d %f * %f,%f\n",i,A[0],bs[i][0],bs[i][1]);
            }
        }
  }
        B += STEP;

        
        __syncthreads();
    } //while( B < Blast );


  if (ibx+id<1000){
    for( int i = 0; i < STEP; i++, C += 1000 )
        if (iby+i<1000){
            C[0] = c[i]; 
            //printf("C[0] = %f ",C[0]);
        }
  }

}	



 __global__ void matrixMultiply( const float *A, const float *B, float* C, int dim )
{
    const int inx = threadIdx.x;
    const int iny = threadIdx.y;
    const int ibx = blockIdx.x * STEP * GROUP;
    const int iby = blockIdx.y * STEP;
    const int id = inx + iny * STEP;

    A += ibx + id;
    B += inx + ( iby + iny) * dim ;
    C += ibx + id  + ( iby * dim );
    
    //printf("[%d,%d] ",blockIdx.x,blockIdx.y);

    float c[STEP] = {0};

    __shared__ float bs[STEP][STEP + 1];
    
//    if (threadIdx.x==0 && threadIdx.y==0)
//                printf("ix:%d iy:%d \n",inx,iny);
    //do
#pragma unroll
    for (int t=0;t<(dim-1)/STEP+1;t++)
    {
#pragma unroll
        for( int i = 0; i < STEP; i += GROUP ){
            if ((t*STEP+inx>=dim) || (iby+iny+i>=dim))
                bs[inx][iny+i] = 0;
            else {
                bs[inx][iny+i] = B[i*dim];
  //              printf("i:%d ix:%d iy:%d \n",i,inx,iny);
            }
        }


        __syncthreads();

  if (ibx+id<dim){
#pragma unroll
        for( int i = 0; i < STEP; i++, A += dim ){
            if (t*STEP+i<dim) {
                saxpy( A[0], &bs[i][0], c ); 
                //printf("i:%d %f * %f,%f\n",i,A[0],bs[i][0],bs[i][1]);
            }
        }
  }
        B += STEP;

        
        __syncthreads();
    } //while( B < Blast );


  if (ibx+id<dim){
    for( int i = 0; i < STEP; i++, C += dim )
        if (iby+i<dim){
            C[0] = c[i]; 
            //printf("C[0] = %f ",C[0]);
        }
  }

}	



  void 
  matrix_multiplication(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, unsigned int sq_dimension)
  {
    int size = sq_dimension * sq_dimension * sizeof(float);
    float *sq_matrix_1_d, *sq_matrix_2_d, *sq_matrix_result_d;
      
    /***************************************************
    1st Part: Allocation of memory on device memory  
    ****************************************************/
  
    /* copy sq_matrix_1 and sq_matrix_2 to device memory */
    hipMalloc((void**) &sq_matrix_1_d, size);
    hipMemcpy(sq_matrix_1_d, sq_matrix_1, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &sq_matrix_2_d, size);
    hipMemcpy(sq_matrix_2_d, sq_matrix_2, size, hipMemcpyHostToDevice);

    /*allocate sq_matrix_result on host */
    hipMalloc((void**) &sq_matrix_result_d, size);
      
    /***************************************************
    2nd Part: Inovke kernel
    ****************************************************/
    dim3 dimGrid((sq_dimension-1)/TILE_WIDTH+1, (sq_dimension-1)/TILE_WIDTH+1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

#ifdef OUTPUT_TIME
    struct timeval tval_before, tval_after, tval_result;
    gettimeofday(&tval_before, NULL);
#endif

    //printf("\n\ndim=%d",sq_dimension);
    dim3 grid( (sq_dimension-1)/STEP/GROUP+1, (sq_dimension-1)/STEP+1 ), 
         threads(STEP, GROUP);
    if (sq_dimension==1024){
        matrixMultiply_1024<<<grid, threads>>>( sq_matrix_2_d, sq_matrix_1_d, sq_matrix_result_d);
    }else if (sq_dimension == 1000){
        matrixMultiply_1000<<<grid, threads>>>( sq_matrix_2_d, sq_matrix_1_d, sq_matrix_result_d);
    } else {
        /*
        printf("\nA = \n");
        for (int i=0;i<sq_dimension;i++){
           for(int j=0;j<sq_dimension;j++)
               printf("%f ",sq_matrix_1[i*sq_dimension+j]);
           printf("\n");
        }

        printf("\nB = \n");
        for (int i=0;i<sq_dimension;i++){
           for(int j=0;j<sq_dimension;j++)
               printf("%f ",sq_matrix_2[i*sq_dimension+j]);
           printf("\n");
        }*/
      matrixMultiply<<<grid, threads>>>(sq_matrix_2_d, sq_matrix_1_d, sq_matrix_result_d, sq_dimension);
    }
//    }else{
//      matrixMultiply<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
//    }
    hipDeviceSynchronize();

#ifdef OUTPUT_TIME
    gettimeofday(&tval_after, NULL);
    timersub(&tval_after, &tval_before, &tval_result);
    printf(" %ld.%06ld\t", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
#endif

    /***************************************************
    3rd Part: Transfer result from device to host
    ****************************************************/
    hipMemcpy(sq_matrix_result, sq_matrix_result_d, size, hipMemcpyDeviceToHost);
    hipFree(sq_matrix_1_d);
    hipFree(sq_matrix_2_d);
    hipFree(sq_matrix_result_d);
       
    //if (sq_dimension==4){
    /*
        printf("\nC = \n");
        for (int i=0;i<sq_dimension;i++){
            for(int j=0;j<sq_dimension;j++){
                printf("%f:",sq_matrix_result[i*sq_dimension+j]);
                float sum = 0;
                for (int k=0;k<sq_dimension;k++)
                    sum+=sq_matrix_1[i*sq_dimension+k]*sq_matrix_2[k*sq_dimension+j];
                printf("%f ",sum);
            }
            printf("\n");
        }
//    }
*/
  }
} // namespace cuda



