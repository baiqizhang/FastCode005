#include "hip/hip_runtime.h"
/*
    Copyright (C) 2011  Abhinav Jauhri (abhinav.jauhri@gmail.com), Carnegie Mellon UniversithreadIdx.y - Silicon Valley 

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANthreadIdx.y; without even the implied warranthreadIdx.y of
    MERCHANTABILIthreadIdx.y or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#define TILE_WIDTH 32
#define TILE_WIDTH_SHIFT 5
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
#include "stdio.h"
#include <sys/time.h>
#define OUTPUT_TIME 
//#define TILE_WIDTH 2

namespace cuda
{
  __global__ void matrixMultiply_1000(float * A, float * B, float * C, int d){
    __shared__ float A_tile[2][TILE_WIDTH][TILE_WIDTH];
    __shared__ float B_tile[2][TILE_WIDTH][TILE_WIDTH];
    int row = (blockIdx.y<<TILE_WIDTH_SHIFT) + threadIdx.y, col = (blockIdx.x<<TILE_WIDTH_SHIFT) + threadIdx.x;
    float sum = 0;
    
    #pragma unroll
    for (int m = 0; m < 30; m+=2) {
      A_tile[0][threadIdx.y][threadIdx.x] = A[row*d + (m<<TILE_WIDTH_SHIFT)+threadIdx.x];
      B_tile[0][threadIdx.y][threadIdx.x] = B[((m<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];


      A_tile[1][threadIdx.y][threadIdx.x] = A[row*d + ((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.x];
      B_tile[1][threadIdx.y][threadIdx.x] = B[(((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];

      __syncthreads();
      #pragma unroll
      for (int k = 0; k < TILE_WIDTH; ++k){
        sum += A_tile[0][threadIdx.y][k] * B_tile[0][k][threadIdx.x];
        sum += A_tile[1][threadIdx.y][k] * B_tile[1][k][threadIdx.x];
      }
      __syncthreads();
    }
    
    #pragma unroll
    for (int m = 30; m < 32; m+=2) {
      if ((m<<TILE_WIDTH_SHIFT)+threadIdx.x < d)
        A_tile[0][threadIdx.y][threadIdx.x] = A[row*d + (m<<TILE_WIDTH_SHIFT)+threadIdx.x];
      else
        A_tile[0][threadIdx.y][threadIdx.x] = 0;

      if ((m<<TILE_WIDTH_SHIFT)+threadIdx.y < d)
        B_tile[0][threadIdx.y][threadIdx.x] = B[((m<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];
      else
        B_tile[0][threadIdx.y][threadIdx.x] = 0;

      if (((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.x < d)
        A_tile[1][threadIdx.y][threadIdx.x] = A[row*d + ((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.x];
      else
        A_tile[1][threadIdx.y][threadIdx.x] = 0;

      if (((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.y < d)
        B_tile[1][threadIdx.y][threadIdx.x] = B[(((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];
      else
        B_tile[1][threadIdx.y][threadIdx.x] = 0;

      __syncthreads();
      #pragma unroll
      for (int k = 0; k < TILE_WIDTH; ++k){
        sum += A_tile[0][threadIdx.y][k] * B_tile[0][k][threadIdx.x];
        sum += A_tile[1][threadIdx.y][k] * B_tile[1][k][threadIdx.x];
      }
      __syncthreads();
    }
    if (row < d && col < d)
      C[row*d + col] = sum;
  }

  __global__ void matrixMultiply_1000_2(float * A, float * B, float * C, int d){
    __shared__ float A_tile[2][TILE_WIDTH][TILE_WIDTH];
    __shared__ float B_tile[2][TILE_WIDTH][TILE_WIDTH];
    int row = (blockIdx.y<<TILE_WIDTH_SHIFT) + threadIdx.y, col = (blockIdx.x<<TILE_WIDTH_SHIFT) + threadIdx.x;
    float sum = 0;
    
    #pragma unroll
    for (int m = 0; m < 30; m+=2) {
      A_tile[0][threadIdx.y][threadIdx.x] = A[row*d + (m<<TILE_WIDTH_SHIFT)+threadIdx.x];
      B_tile[0][threadIdx.y][threadIdx.x] = B[((m<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];

      if (m!=0){
        #pragma unroll
        for (int k = 0; k < TILE_WIDTH; ++k){
          sum += A_tile[1][threadIdx.y][k] * B_tile[1][k][threadIdx.x];
        }
      }
      __syncthreads();
      
      #pragma unroll
      for (int k = 0; k < TILE_WIDTH; ++k){
        sum += A_tile[0][threadIdx.y][k] * B_tile[0][k][threadIdx.x];
      }
      
      A_tile[1][threadIdx.y][threadIdx.x] = A[row*d + ((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.x];
      B_tile[1][threadIdx.y][threadIdx.x] = B[(((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];

      __syncthreads();
    }
    
    #pragma unroll
    for (int m = 30; m < 32; m+=2) {
      if ((m<<TILE_WIDTH_SHIFT)+threadIdx.x < d)
        A_tile[0][threadIdx.y][threadIdx.x] = A[row*d + (m<<TILE_WIDTH_SHIFT)+threadIdx.x];
      else
        A_tile[0][threadIdx.y][threadIdx.x] = 0;

      if ((m<<TILE_WIDTH_SHIFT)+threadIdx.y < d)
        B_tile[0][threadIdx.y][threadIdx.x] = B[((m<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];
      else
        B_tile[0][threadIdx.y][threadIdx.x] = 0;

      #pragma unroll
      for (int k = 0; k < TILE_WIDTH; ++k){
        sum += A_tile[1][threadIdx.y][k] * B_tile[1][k][threadIdx.x];
      }

      __syncthreads();
      #pragma unroll
      for (int k = 0; k < TILE_WIDTH; ++k){
        sum += A_tile[0][threadIdx.y][k] * B_tile[0][k][threadIdx.x];
      }
      if (((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.x < d)
        A_tile[1][threadIdx.y][threadIdx.x] = A[row*d + ((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.x];
      else
        A_tile[1][threadIdx.y][threadIdx.x] = 0;

      if (((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.y < d)
        B_tile[1][threadIdx.y][threadIdx.x] = B[(((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];
      else
        B_tile[1][threadIdx.y][threadIdx.x] = 0;
      __syncthreads();
    }
    #pragma unroll
    for (int k = 0; k < TILE_WIDTH; ++k){
        sum += A_tile[1][threadIdx.y][k] * B_tile[1][k][threadIdx.x];
    }
    if (row < d && col < d)
      C[row*d + col] = sum;
  }






  __global__ void matrixMultiply_1024_32thread(float * A, float * B, float * C, int d){
    __shared__ float A_tile[2][TILE_WIDTH][TILE_WIDTH];
    __shared__ float B_tile[2][TILE_WIDTH][TILE_WIDTH];
    
    int row = (blockIdx.y<<TILE_WIDTH_SHIFT) + threadIdx.y, col = (blockIdx.x<<TILE_WIDTH_SHIFT) + threadIdx.x;
    float sum = 0;

    #pragma unroll
    for (int m = 0; m < TILE_WIDTH; m+=2) {
      if (m!=0){
        #pragma unroll
        for (int k = 0; k < TILE_WIDTH; ++k){
          sum += A_tile[1][k][threadIdx.y] * B_tile[1][k][threadIdx.x];
          //sum += A_tile[1][threadIdx.y][k] * B_tile[1][k][threadIdx.x];
        }
      }
      
      A_tile[0][threadIdx.x][threadIdx.y] = A[row*d + (m<<TILE_WIDTH_SHIFT)+threadIdx.x];
      B_tile[0][threadIdx.y][threadIdx.x] = //B[col*d + (m<<TILE_WIDTH_SHIFT)+threadIdx.y];
                                            B[((m<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];

      __syncthreads();

      #pragma unroll
      for (int k = 0; k < TILE_WIDTH; ++k){
        sum += A_tile[0][k][threadIdx.y] * B_tile[0][k][threadIdx.x];
        //sum += A_tile[1][threadIdx.y][k] * B_tile[1][k][threadIdx.x];
      }
      A_tile[1][threadIdx.x][threadIdx.y] = A[row*d + ((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.x];
      B_tile[1][threadIdx.y][threadIdx.x] = //B[col*d + ((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.y];
                                            B[(((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];
      __syncthreads();
    }
    
    #pragma unroll
    for (int k = 0; k < TILE_WIDTH; ++k){
        sum += A_tile[1][k][threadIdx.y] * B_tile[1][k][threadIdx.x];
    }

    C[row*d + col] = sum;
  }
  






  __global__ void matrixMultiply_1024(float * A, float * B, float * C, int d){
    __shared__ float A_tile[2][TILE_WIDTH][TILE_WIDTH];
    //__shared__ float test[1];
    __shared__ float B_tile[2][TILE_WIDTH][TILE_WIDTH];
    int row = (blockIdx.y<<TILE_WIDTH_SHIFT) + threadIdx.y, col = (blockIdx.x<<TILE_WIDTH_SHIFT) + threadIdx.x;
    float sum = 0; 
    #pragma unroll
    for (int m = 0; m < 32; m+=2) {
      if (m!=0){
        #pragma unroll
        for (int k = 0; k < TILE_WIDTH; ++k){
          sum += A_tile[1][threadIdx.y][k] * B_tile[1][k][threadIdx.x];
          //sum += A_tile[1][threadIdx.y][k] * B_tile[1][k][threadIdx.x];
        }
      }
      
      A_tile[0][threadIdx.y][threadIdx.x] = A[row*d + (m<<TILE_WIDTH_SHIFT)+threadIdx.x];
      B_tile[0][threadIdx.y][threadIdx.x] = //B[col*d + (m<<TILE_WIDTH_SHIFT)+threadIdx.y];
                                            B[((m<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];

      __syncthreads();

      #pragma unroll
      for (int k = 0; k < TILE_WIDTH; ++k){
        sum += A_tile[0][threadIdx.y][k] * B_tile[0][k][threadIdx.x];
        //sum += A_tile[1][threadIdx.y][k] * B_tile[1][k][threadIdx.x];
      }
      A_tile[1][threadIdx.y][threadIdx.x] = A[row*d + ((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.x];
      B_tile[1][threadIdx.y][threadIdx.x] = //B[col*d + ((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.y];
                                            B[(((m+1)<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];
      __syncthreads();
    }
    
    #pragma unroll
    for (int k = 0; k < TILE_WIDTH; ++k){
        sum += A_tile[1][threadIdx.y][k] * B_tile[1][k][threadIdx.x];
    }

    C[row*d + col] = sum;
  }
  










  // Compute C = A * B
  __global__ void matrixMultiply(float * A, float * B, float * C, int d){
    __shared__ float A_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ float B_tile[TILE_WIDTH][TILE_WIDTH];
    int row = (blockIdx.y<<TILE_WIDTH_SHIFT) + threadIdx.y, col = (blockIdx.x<<TILE_WIDTH_SHIFT) + threadIdx.x;
    float sum = 0;
    
    #pragma unroll
    for (int m = 0; m < (d-1)/TILE_WIDTH+1; ++m) {
      if ((m<<TILE_WIDTH_SHIFT)+threadIdx.x < d)
        A_tile[threadIdx.y][threadIdx.x] = A[row*d + (m<<TILE_WIDTH_SHIFT)+threadIdx.x];
      else
        A_tile[threadIdx.y][threadIdx.x] = 0;

      if ((m<<TILE_WIDTH_SHIFT)+threadIdx.y < d)
        B_tile[threadIdx.y][threadIdx.x] = B[((m<<TILE_WIDTH_SHIFT)+threadIdx.y)*d+col];
      else
        B_tile[threadIdx.y][threadIdx.x] = 0;

      __syncthreads();
      #pragma unroll
      for (int k = 0; k < TILE_WIDTH; ++k)
        sum += A_tile[threadIdx.y][k] * B_tile[k][threadIdx.x];
      __syncthreads();
    }
    if (row < d && col < d)
      C[row*d + col] = sum;
  }




__device__ void saxpy( float a, float *b, float *c )
{
    c[0] += a*b[0];
    c[1] += a*b[1];
    c[2] += a*b[2];
    c[3] += a*b[3];
    c[4] += a*b[4];
    c[5] += a*b[5];
    c[6] += a*b[6];
    c[7] += a*b[7];
    c[8] += a*b[8];
    c[9] += a*b[9];
    c[10] += a*b[10];
    c[11] += a*b[11];
    c[12] += a*b[12];
    c[13] += a*b[13];
    c[14] += a*b[14];
    c[15] += a*b[15];
}


  __global__ void matrixMultiply_1024_2(const float * A, const float * B,float * C, int dim){

    const int inx = threadIdx.x;
    const int iny = threadIdx.y;
    const int ibx = blockIdx.x * 64;
    const int iby = blockIdx.y * 16;
    const int id = inx + iny*16;

    __shared__ float as[16][17];
    float c[16]={0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

//    A += (blockIdx.x*64) + threadIdx.x + threadIdx.y*16;
//    B += threadIdx.x + (blockIdx.y*16 + threadIdx.y)*dim;
//    C += blockIdx.x*64 + threadIdx.x+(threadIdx.y+blockIdx.y*dim)*16;
    B += ibx + id;
    A += inx + __mul24( iby + iny, dim );
    C += ibx + id  + __mul24( iby, dim );
    
    const float *Alast = A+dim;

    do {
#pragma unroll
        for( int i = 0; i < 16; i += 4 )
            as[inx][iny+i] = A[i*dim];
        __syncthreads();

#pragma unroll
        for( int i = 0; i < 16; i++, B += dim )
            saxpy( B[0], &as[i][0], c ); 

        A += 16;
        __syncthreads();
    } while( A < Alast );

    for( int i = 0; i < 16; i++, C += dim )
        C[0] = c[i] + C[0];
}






  // Compute C = A * B
  __global__ void preProcess(float * A, float * B, float * C, int d){
    int row = (blockIdx.y<<TILE_WIDTH_SHIFT) + threadIdx.y, col = (blockIdx.x<<TILE_WIDTH_SHIFT) + threadIdx.x;
     if (row > col || row>=d || col>=d )
         return;
     float t = C[row*d+col];
     C[row*d+col] = C[col*d+row];
     C[col*d+row] = t;
  }

  void 
  matrix_multiplication(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, unsigned int sq_dimension)
  {
    int size = sq_dimension * sq_dimension * sizeof(float);
    float *sq_matrix_1_d, *sq_matrix_2_d, *sq_matrix_result_d;
      
    /***************************************************
    1st Part: Allocation of memory on device memory  
    ****************************************************/
  
    /* copy sq_matrix_1 and sq_matrix_2 to device memory */
    hipMalloc((void**) &sq_matrix_1_d, size);
    hipMemcpy(sq_matrix_1_d, sq_matrix_1, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &sq_matrix_2_d, size);
    hipMemcpy(sq_matrix_2_d, sq_matrix_2, size, hipMemcpyHostToDevice);

    /*allocate sq_matrix_result on host */
    hipMalloc((void**) &sq_matrix_result_d, size);
      
    /***************************************************
    2nd Part: Inovke kernel
    ****************************************************/
    dim3 dimGrid((sq_dimension-1)/TILE_WIDTH+1, (sq_dimension-1)/TILE_WIDTH+1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

#ifdef OUTPUT_TIME
    struct timeval tval_before, tval_after, tval_result;
    gettimeofday(&tval_before, NULL);
#endif

    if (sq_dimension==1024){
      //preProcess<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
      //matrixMultiply_1024<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
        dim3 grid( 1024/64, 1024/16 ), threads(16, 4);
        matrixMultiply_1024_2<<<grid, threads>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
    }else if (sq_dimension == 1000){
      matrixMultiply_1000_2<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
    }else{
      matrixMultiply<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
    }
    hipDeviceSynchronize();

#ifdef OUTPUT_TIME
    gettimeofday(&tval_after, NULL);
    timersub(&tval_after, &tval_before, &tval_result);
    printf(" %ld.%06ld\t", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
#endif

    /***************************************************
    3rd Part: Transfer result from device to host
    ****************************************************/
    hipMemcpy(sq_matrix_result, sq_matrix_result_d, size, hipMemcpyDeviceToHost);
    hipFree(sq_matrix_1_d);
    hipFree(sq_matrix_2_d);
    hipFree(sq_matrix_result_d);
  }
} // namespace cuda

